#include "hip/hip_runtime.h"
/*
Maximal independent set code

Copyright 2022 Martin Burtscher

Redistribution in source or binary form, with or without modification, is not
permitted. Use in source or binary form, with or without modification, is only
permitted for academic use in CS 4380 and CS 5351 at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/*
  Worked by:
    - Chuong Dinh Vu
    - Noah del Angel
*/


#include <cstdlib>
#include <cstdio>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "ECLgraph.h"


static const unsigned char included = 0;
static const unsigned char excluded = 1;
static const unsigned char undecided = 2;

static const int ThreadsPerBlock = 512;



// https://stackoverflow.com/questions/664014/what-integer-hash-function-are-good-that-accepts-an-integer-hash-key
static __device__ unsigned int hash(unsigned int val)
{
  val = ((val >> 16) ^ val) * 0x45D9F3B;
  val = ((val >> 16) ^ val) * 0x45D9F3B;
  return (val >> 16) ^ val;
}

static __global__ void init( const  ECLgraph  g, volatile unsigned int* const priority, volatile unsigned char* const status ) 
{
  const int v = threadIdx.x + blockIdx.x * blockDim.x;
  
  // initialize arrays
  if (v < g.nodes) {
    status[v] = undecided;
    priority[v] = hash(v + 712813837);
  }
}

static __global__ void mis(const  ECLgraph  g,  volatile  unsigned char* const status, const unsigned int* const priority, volatile bool* const repeat)
{
  const int v = threadIdx.x + blockIdx.x * blockDim.x;

  // go over all nodes
  if ( v < g.nodes ) {
    if (status[v] == undecided) {
      int i = g.nindex[v];
      // try to find a neighbor whose priority is higher
      while ((i < g.nindex[v + 1]) && ((status[g.nlist[i]] == excluded) || (priority[v] > priority[g.nlist[i]]) || ((priority[v] == priority[g.nlist[i]]) && (v < g.nlist[i])))) {
        i++;
      }
      if (i < g.nindex[v + 1]) {
        // found such a neighbor -> status of v still unknown
        *repeat = true;
      } else {
        // no such neighbor -> all neighbors are "excluded" and v is "included"
        status[v] = included;
        for (int i = g.nindex[v]; i < g.nindex[v + 1]; i++) {
          status[g.nlist[i]] = excluded;
        }
      }
    }
  }
}

static void CheckCuda(const int line)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d on line %d: %s\n", e, line, hipGetErrorString(e));
    exit(-1);
  }
}


int main(int argc, char* argv [])
{
  printf("Maximal independent set v1.9\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "USAGE: %s input_file\n", argv[0]); exit(-1);}

  // read input
  ECLgraph g = readECLgraph(argv[1]);
  printf("input: %s\n", argv[1]);
  printf("nodes: %d\n", g.nodes);
  printf("edges: %d\n", g.edges);

  // allocate arrays
  unsigned char* const status = new unsigned char [g.nodes];
  unsigned int* const priority = new unsigned int [g.nodes];

  // Allocate repeate flag
  bool repeat;

  // alloace GPU items
  unsigned char* const d_status = new unsigned char [g.nodes];
  bool* d_repeat;

  hipMalloc((void **) &d_status, sizeof(unsigned char) * g.nodes);
  CheckCuda(__LINE__);
  hipMalloc((void **) &d_repeat, sizeof(bool));
  CheckCuda(__LINE__);

  // Allocate g graph to GPU
  ECLgraph d_g = g; 
  hipMalloc((void **)&d_g.nindex, sizeof(int) * (g.nodes + 1)); 
  CheckCuda(__LINE__); 
  hipMalloc((void **)&d_g.nlist, sizeof(int) * g.edges); 
  CheckCuda(__LINE__); 
  hipMemcpy(d_g.nindex, g.nindex, sizeof(int) * (g.nodes + 1), hipMemcpyHostToDevice); 
  CheckCuda(__LINE__); 
  hipMemcpy(d_g.nlist, g.nlist, sizeof(int) * g.edges, hipMemcpyHostToDevice); 
  CheckCuda(__LINE__);


  // start time
  timeval beg, end;
  gettimeofday(&beg, NULL);

  // execute timed code
  init <<<( g.nodes + ThreadsPerBlock - 1 ) / ThreadsPerBlock, ThreadsPerBlock>>>(d_g, priority, d_status);

  // repeat until all nodes' statuses have been decided
  do {
    repeat = false;
    hipMemcpy(d_repeat, &repeat, sizeof(bool), hipMemcpyHostToDevice);


    mis<<<( g.nodes + ThreadsPerBlock - 1 ) / ThreadsPerBlock, ThreadsPerBlock>>>(d_g, d_status, priority, d_repeat);
    hipDeviceSynchronize();
    
    // return the repeat to the host
    hipMemcpy(&repeat, d_repeat, sizeof(bool), hipMemcpyDeviceToHost);
  } while (repeat);

  // end time
  gettimeofday(&end, NULL);
  CheckCuda(__LINE__);

  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);


  // Return status array from GPU
  hipMemcpy(status, d_status, sizeof(unsigned char) * g.nodes, hipMemcpyDeviceToHost);
  CheckCuda(__LINE__);

  // determine and print set size
  int cnt = 0;
  for (int v = 0; v < g.nodes; v++) {
    if (status[v] == included) cnt++;
  }
  printf("elements in set: %d (%.1f%%)\n", cnt, 100.0 * cnt / g.nodes);

  // verify result
  for (int v = 0; v < g.nodes; v++) {
    if ((status[v] != included) && (status[v] != excluded)) {fprintf(stderr, "ERROR: found undecided node\n"); exit(-1);}
    if (status[v] == included) {
      for (int i = g.nindex[v]; i < g.nindex[v + 1]; i++) {
        if (status[g.nlist[i]] == included) {fprintf(stderr, "ERROR: found adjacent nodes in MIS\n"); exit(-1);}
      }
    } else {
      bool flag = true;
      for (int i = g.nindex[v]; i < g.nindex[v + 1]; i++) {
        if (status[g.nlist[i]] == included) {
          flag = false;
          break;
        }
      }
      if (flag) {fprintf(stderr, "ERROR: set is not maximal\n"); exit(-1);}
    }
  }
  printf("verification passed\n");

  // clean up
  freeECLgraph(g);
  delete [] priority;
  delete [] status;
  hipFree(d_g.nindex);
  hipFree(d_g.nlist);
  hipFree(d_status);
  hipFree(d_repeat);
  return 0;
}
