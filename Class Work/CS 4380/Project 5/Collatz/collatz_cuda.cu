/*
Collatz code

Copyright 2022 Martin Burtscher

Redistribution in source or binary form, with or without modification, is not
permitted. Use in source or binary form, with or without modification, is only
permitted for academic use in CS 4380 and CS 5351 at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/*
  Worked by:
    - Chuong Dinh Vu
    - Noah del Angel
*/


#include <cstdio>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <sys/time.h>


static const int ThreadsPerBlock = 512;


static __global__ void collatz(const long start, const long top, int* const maxlen)
{
  // Determine what thread we are
  const long idx = start + (threadIdx.x + blockIdx.x * (long)blockDim.x) * 2;

  // compute sequence lengths
  if ( idx < top )
  {
    long val = idx;
    int len = 1;
    do {
      len++;
      if ((val % 2) != 0) {
        val = 3 * val + 1;  // odd
      } else {
        val = val / 2;  // even
      }
    } while (val != 1);
    atomicMax(maxlen, len);
  }
  
}

static void CheckCuda(const int line)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d on line %d; %s\n", e, line, hipGetErrorString(e));
    exit(-1);
  }
}


int main(int argc, char* argv [])
{
  printf("Collatz v1.8\n");

  // check command line
  if (argc != 3) {fprintf(stderr, "USAGE: %s start_value upper_bound\n", argv[0]); exit(-1);}
  const long start = atol(argv[1]);
  const long top = atol(argv[2]);
  printf("start value: %ld\n", start);
  printf("upper bound: %ld\n", top);

  // allocate variable on CPU
  int maxlen = 0;

  // allocate variable on GPU
  int* const d_maxlen = new int;

  hipMalloc((void **) &d_maxlen, sizeof(int));
  CheckCuda(__LINE__);

  // initialize the values
  hipMemcpy(d_maxlen, &maxlen, sizeof(int), hipMemcpyHostToDevice);
  CheckCuda(__LINE__);

  // start time
  timeval beg, end;
  gettimeofday(&beg, NULL);

  // execute timed code
  collatz<<<((top - start + 1)/2 + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(start, top, d_maxlen);
  hipDeviceSynchronize();  // wait for kernel to finish

  // end time
  gettimeofday(&end, NULL);
  CheckCuda(__LINE__);
  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);

  // get result from GPU 
  hipMemcpy(&maxlen, d_maxlen, sizeof(int), hipMemcpyDeviceToHost);
  CheckCuda(__LINE__);

  // print result
  printf("maximum length: %d\n", maxlen);

  // clean up
  hipFree(d_maxlen);
  return 0;
}
