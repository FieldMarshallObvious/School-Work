/*
Vector addition code

Copyright 2022 Martin Burtscher

Redistribution in source or binary form, with or without modification, is not
permitted. Use in source or binary form, with or without modification, is only
permitted for academic use in CS 4380 and CS 5351 at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>


static const int ThreadsPerBlock = 512;


static __global__ void vadd(const int* const a, const int* const b, int* const c, const int elems)
{
  // perform vector addition
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < elems) {
    c[i] = a[i] + b[i];
  }
}


static void CheckCuda(const int line)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d on line %d: %s\n", e, line, hipGetErrorString(e));
    exit(-1);
  }
}


int main(int argc, char* argv [])
{
  printf("Vector addition v1.6\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "USAGE: %s vector_elements\n", argv[0]); exit(-1);}
  const int elems = atoi(argv[1]);
  if (elems < 1) {fprintf(stderr, "ERROR: vector_elements must be at least 1\n"); exit(-1);}
  printf("vector elements: %d\n", elems);

  // allocate vectors
  int* const a = new int [elems];
  int* const b = new int [elems];
  int* const c = new int [elems];

  // initialize vectors
  for (int i = 0; i < elems; i++) a[i] = i;
  for (int i = 0; i < elems; i++) b[i] = elems - i;
  for (int i = 0; i < elems; i++) c[i] = -1;

  // allocate vectors on GPU
  int* d_a;
  int* d_b;
  int* d_c;
  hipMalloc((void **)&d_a, sizeof(int) * elems);
  CheckCuda(__LINE__);
  hipMalloc((void **)&d_b, sizeof(int) * elems);
  CheckCuda(__LINE__);
  hipMalloc((void **)&d_c, sizeof(int) * elems);
  CheckCuda(__LINE__);

  // initialize vectors on GPU
  hipMemcpy(d_a, a, sizeof(int) * elems, hipMemcpyHostToDevice);
  CheckCuda(__LINE__);
  hipMemcpy(d_b, b, sizeof(int) * elems, hipMemcpyHostToDevice);
  CheckCuda(__LINE__);
  hipMemcpy(d_c, c, sizeof(int) * elems, hipMemcpyHostToDevice);  // optional
  CheckCuda(__LINE__);

  // start time
  timeval beg, end;
  gettimeofday(&beg, NULL);

  // execute timed code
  vadd<<<(elems + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_a, d_b, d_c, elems);
  hipDeviceSynchronize();  // wait for kernel to finish

  // end time
  gettimeofday(&end, NULL);
  CheckCuda(__LINE__);
  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);

  // get result from GPU
  hipMemcpy(c, d_c, sizeof(int) * elems, hipMemcpyDeviceToHost);
  CheckCuda(__LINE__);

  // verify result
  for (int i = 0; i < elems; i++) {
    if (c[i] != elems) {fprintf(stderr, "ERROR: incorrect result\n"); exit(-1);}
  }
  printf("verification passed\n");

  // clean up
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  delete [] a;
  delete [] b;
  delete [] c;
  return 0;
}
