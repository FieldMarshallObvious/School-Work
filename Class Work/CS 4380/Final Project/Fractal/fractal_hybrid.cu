/*
Fractal code

Copyright 2022 Martin Burtscher

Redistribution in source or binary form, with or without modification, is not
permitted. Use in source or binary form, with or without modification, is only
permitted for academic use in CS 4380 and CS 5351 at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>


static const int ThreadsPerBlock = 512;


static __global__ void fractal(const int start_frame, const int stop_frame, const int frames, const int width, unsigned char* const pic)
{
  // todo: compute the pixels of all frames in the range
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int col = i % width;
  const int row = ( i / width ) % width;
  const int frame = i / ( width * width ) + start_frame;

  const double Delta = 0.004937716;
  const double xMid = -0.664689302;
  const double yMid =  0.355561972;

    // compute pixels of each frame
  if( i < ( stop_frame *  width * width )){  // frames
    const double delta = Delta * (1.5 + cos(2.0 * M_PI * frame / frames));
    const double xMin = xMid - delta;
    const double yMin = yMid - delta;
    const double dw = 2.0 * delta / width;
    const double cy = yMin + row * dw;
    const double cx = xMin + col * dw;
    double x = cx;
    double y = cy;
    double x2, y2;
    int count = 256;
    do {
      x2 = x * x;
      y2 = y * y;
      y = 2.0 * x * y + cy;
      x = x2 - y2 + cx;
      count--;
    } while ((count > 0) && ((x2 + y2) < 4.0));
    pic[((frame-start_frame) * width * width + row * width + col)] = (unsigned char)count;
  }
}


static void CheckCuda(const int line)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d on line %d: %s\n", e, line, hipGetErrorString(e));
    exit(-1);
  }
}


unsigned char* GPU_Init(const int gpu_frames, const int width)
{
  unsigned char* d_pic;
  hipMalloc((void **)&d_pic, gpu_frames * width * width * sizeof(unsigned char));
  CheckCuda(__LINE__);
  return d_pic;
}


void GPU_Exec(const int start_frame, const int stop_frame, const int frames, const int width, unsigned char* d_pic)
{
  // todo: launch the kernel with just the right number of blocks and ThreadsPerBlock threads per block and do nothing else
  fractal<<<(((stop_frame - start_frame) * width * width) + ThreadsPerBlock - 1 ) / ThreadsPerBlock , ThreadsPerBlock>>>(start_frame, stop_frame, frames, width, d_pic);
}


void GPU_Fini(const int gpu_frames, const int width, unsigned char* pic, unsigned char* d_pic)
{
  // todo: copy the result from the device to the host and free the device memory
  hipMemcpy(pic, d_pic, gpu_frames * width * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
  CheckCuda(__LINE__);
  hipFree(d_pic);
  CheckCuda(__LINE__);
}
