/*
Collatz code

Copyright 2022 Martin Burtscher

Redistribution in source or binary form, with or without modification, is not
permitted. Use in source or binary form, with or without modification, is only
permitted for academic use in CS 4380 and CS 5351 at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <cstdio>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <sys/time.h>


static const int ThreadsPerBlock = 512;


static __global__ void collatz(const long start, const long top, const long increment, int* const maxlen)
{
  // Determine what thread we are
  const long idx = start + (threadIdx.x + blockIdx.x * (long)blockDim.x) * increment;

  // compute sequence lengths
  if ( idx < top )
  {
    long val = idx;
    int len = 1;
    do {
      len++;
      if ((val % 2) != 0) {
        val = 3 * val + 1;  // odd
      } else {
        val = val / 2;  // even
      }
    } while (val != 1);
    atomicMax(maxlen, len);
  }
}


static void CheckCuda(const int line)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d on line %d: %s\n", e, line, hipGetErrorString(e));
    exit(-1);
  }
}


int main(int argc, char* argv [])
{
  printf("Collatz v1.8\n");

  // check command line
  if (argc != 4) {fprintf(stderr, "USAGE: %s start_value upper_bound number_of_GPUs\n", argv[0]); exit(-1);}
  const long start = atol(argv[1]);
  const long top = atol(argv[2]);
  const int gpus = atoi(argv[3]);
  if ((gpus < 1) || (gpus > 4)) {fprintf(stderr, "ERROR: number of GPUs must be between 1 and 4\n"); exit(-1);}
  printf("start value: %ld\n", start);
  printf("upper bound: %ld\n", top);
  printf("number of GPUs: %d\n", gpus);

  // allocate and initialize GPU memory
  int maxlen = 0;
  int* d_maxlen [4];
  for (int d = 0; d < gpus; d++) {
    hipSetDevice(d);
    hipMalloc((void **)&d_maxlen[d], sizeof(int));
    CheckCuda(__LINE__);
  }
  // todo: set the maximum length on each GPU to zero by copying maxlen to the GPU
  for (int d = 0; d < gpus; d++) {
    hipSetDevice(d);
    hipMemcpy(d_maxlen[d], &maxlen, sizeof(int), hipMemcpyHostToDevice);
    CheckCuda(__LINE__);
  }

  // start time
  timeval beg, end;
  gettimeofday(&beg, NULL);

  // execute timed code
  for (int d = 0; d < gpus; d++) {
    hipSetDevice(d);
    const long begin = start + d * (((top - start + 1)/2 + ThreadsPerBlock - 1) / ThreadsPerBlock);
    const long incr = 2 * gpus;
    collatz<<< ((top - start + 1)/2 + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(begin, top, incr, d_maxlen[d]);
  }
  for (int d = 0; d < gpus; d++) {
    hipSetDevice(d);
    hipDeviceSynchronize();  // wait for kernel to finish
  }

  // end time
  gettimeofday(&end, NULL);
  CheckCuda(__LINE__);
  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);

  // todo: get the result from each GPU and reduce the values on the CPU into a final global result
  for( int d = 0; d < gpus; d++ ) {
    int tempMax = 0;
    
    hipSetDevice(d);
    hipMemcpy(&tempMax, d_maxlen[d], sizeof(int), hipMemcpyDeviceToHost);
    CheckCuda(__LINE__);

    // Only assign CPU maxlen if the one from 
    // the gpu is greater 
    if( tempMax > maxlen ) {
      maxlen = tempMax;
    }
  }
  printf("maximum length: %d\n", maxlen);

  for (int d = 0; d < gpus; d++) {
    hipSetDevice(d);
    hipFree(d_maxlen[d]);
    CheckCuda(__LINE__);
  }
  return 0;
}
